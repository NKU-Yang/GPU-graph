#include "hip/hip_runtime.h"
#include"Graph.h"

extern "C"{
__global__
void simpleBFS_CUDA(uint N, uint level, EDGE_POINTER_TYPE* nodePointers, uint* edgeArr, uint* distance, bool* visited){
    int thid = blockIdx.x * blockDim.x + threadIdx.x;
    if(thid<N && distance[thid]==level)
    {
        int u= thid;
        for(uint i=0;i<nodePointers[u+1]-nodePointers[u];i++)
        {
            uint v = edgeArr[u+i];
            if(visited[v]==false&&(distance[v]==-1||distance[v]>level+1))
            {
                distance[v]=level+1;
            }
        }
    }
}

void BFS_CUDA(Graph<uint> G, uint source, uint* distance, bool* visited)
    {
        EDGE_POINTER_TYPE *d_nodePointers;
        uint *d_edgeArr;
        uint nodeSize = G.vertexArrSize;
        uint edgeSize = G.edgeArrSize;
        uint* d_distacne;
        bool* d_visited;
        uint* d_cur_frointers;
        queue<uint> frointers;
        
        ///initial CUDA
        //malloc mem
        hipMalloc(&d_nodePointers, sizeof(EDGE_POINTER_TYPE)*nodeSize);
        checkError("hipMalloc d_nodePointers ");
        hipMalloc(&d_edgeArr, sizeof(uint)*edgeSize);
        checkError("hipMalloc d_edgeArr ");
        hipMalloc(&d_distacne, sizeof(uint)*nodeSize);
        checkError("hipMalloc d_distance ");
        hipMalloc(&d_visited, sizeof(bool)*nodeSize);
        checkError("hipMalloc d_visited ");
        hipMalloc(&d_cur_frointers, sizeof(uint)*nodeSize);
        //copy init data
        gpuErrorcheck(hipMemcpy(d_nodePointers, G.nodePointers, sizeof(EDGE_POINTER_TYPE)*nodeSize, hipMemcpyHostToDevice));
        gpuErrorcheck(hipMemcpy(d_edgeArr, G.edgeArray, sizeof(uint)*edgeSize, hipMemcpyHostToDevice));
        gpuErrorcheck(hipMemcpy(d_distacne, distance, sizeof(uint)*nodeSize, hipMemcpyHostToDevice));
        gpuErrorcheck(hipMemcpy(d_visited, visited, sizeof(bool)*nodeSize, hipMemcpyHostToDevice));
        
        frointers.push(source);
        visited[source]=true;
        distance[source]=0;
        dim3 grid=(56,1,1);
        dim3 block=(1024,1,1);
        uint level=0;
        while(!frointers.empty())
        {
            uint u=frointers.front();
            //bfs_kernel
            simpleBFS_CUDA<<<grid, block>>>(nodeSize, level, d_nodePointers, d_edgeArr, d_distacne, d_visited);
            frointers.pop();
        }
    }
}
